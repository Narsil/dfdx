#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

template<typename F>
struct ScalarAddKernelOp {
    F scalar;
};

#if __CUDA_ARCH__ >= 530
UNARY_OP(__half, sadd_fwd_f16, sadd_bwd_f16, ScalarAddKernelOp<__half>,
    x + op.scalar,
    1.0);
#endif

UNARY_OP(float, sadd_fwd_f32, sadd_bwd_f32, ScalarAddKernelOp<float>,
    x + op.scalar,
    1.0);

UNARY_OP(double, sadd_fwd_f64, sadd_bwd_f64, ScalarAddKernelOp<double>,
    x + op.scalar,
    1.0);
    
