#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

template<typename F>
struct ScalarMulKernelOp {
    F scalar;
};

#if __CUDA_ARCH__ >= 530
UNARY_OP(__half, smul_fwd_f16, smul_bwd_f16, ScalarMulKernelOp<__half>,
    x * op.scalar,
    op.scalar);
#endif

UNARY_OP(float, smul_fwd_f32, smul_bwd_f32, ScalarMulKernelOp<float>,
    x * op.scalar,
    op.scalar);

UNARY_OP(double, smul_fwd_f64, smul_bwd_f64, ScalarMulKernelOp<double>,
    x * op.scalar,
    op.scalar);
    
